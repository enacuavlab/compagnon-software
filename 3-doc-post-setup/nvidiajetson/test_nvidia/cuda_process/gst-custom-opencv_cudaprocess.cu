#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include "cudaEGL.h"

#if defined(__cplusplus)
extern "C" void Handle_EGLImage (EGLImageKHR image);
extern "C" {
#endif

typedef enum {
  COLOR_FORMAT_Y8 = 0,
  COLOR_FORMAT_U8_V8,
  COLOR_FORMAT_RGBA,
  COLOR_FORMAT_NONE
} ColorFormat;

typedef struct {
  void (*fGPUProcess) (EGLImageKHR image, void ** userPtr);
  void (*fPreProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
  void (*fPostProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
} CustomerFunction;

void init (CustomerFunction * pFuncs);

#if defined(__cplusplus)
}
#endif

static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
  /* add your custom pre-process here */
}

static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
  /* add your custom post-process here */
}

static void cv_process(void *pdata, int32_t width, int32_t height)
{
//  cv::cuda::GpuMat d_mat(height, width, CV_8UC4, pdata);
}

static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }

  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_ABGR) {
	cv_process(eglFrame.frame.pPitch[0], eglFrame.width, eglFrame.height);
    } else {
	printf ("Invalid eglcolorformat for opencv\n");
	std::cout<<eglFrame.eglColorFormat << ", " <<CU_EGL_COLOR_FORMAT_RGBA<<std::endl;
    }
  }
  else {
     printf ("Invalid frame type for opencv\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }
}

extern "C" void
init (CustomerFunction * pFuncs)
{
  pFuncs->fPreProcess = pre_process;
  pFuncs->fGPUProcess = gpu_process;
  pFuncs->fPostProcess = post_process;
}
