#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#include "opencv2/core.hpp"
#include "opencv2/calib3d.hpp"
#include "opencv2/cudawarping.hpp" 

#include "cudaEGL.h"

#if defined(__cplusplus)
extern "C" void Handle_EGLImage (EGLImageKHR image);
extern "C" {
#endif

typedef enum {
  COLOR_FORMAT_Y8 = 0,
  COLOR_FORMAT_U8_V8,
  COLOR_FORMAT_RGBA,
  COLOR_FORMAT_NONE
} ColorFormat;

typedef struct {
  void (*fGPUProcess) (EGLImageKHR image, void ** userPtr);
  void (*fPreProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
  void (*fPostProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
} CustomerFunction;

void init (CustomerFunction * pFuncs);

#if defined(__cplusplus)
}
#endif


static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
   printf ("pre-process %dx%d size %d\n", *swidth, *sheight, *smemsize); 
}

static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
   printf ("post-process %dx%d size %d\n", *swidth, *sheight, *smemsize); 
}



static cv::cuda::GpuMat gpu_xmap, gpu_ymap;

static void cv_process_RGBA(void *pdata, int32_t width, int32_t height)
{
    cv::cuda::GpuMat d_Mat_RGBA(height, width, CV_8UC4, pdata);
    cv::cuda::GpuMat d_Mat_RGBA_Src;
    d_Mat_RGBA.copyTo(d_Mat_RGBA_Src); // cannot avoid one copy
    cv::cuda::remap(d_Mat_RGBA_Src, d_Mat_RGBA, gpu_xmap, gpu_ymap, cv::INTER_CUBIC, cv::BORDER_CONSTANT, cv::Scalar(0.f, 0.f, 0.f, 0.f));

    // Check
    if(d_Mat_RGBA.data != pdata)
	std::cerr << "Error reallocated buffer for d_Mat_RGBA" << std::endl;
}

static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);

  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }

  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_ABGR) {
 	cv_process_RGBA(eglFrame.frame.pPitch[0], eglFrame.width, eglFrame.height);
    } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
      printf ("Invalid eglcolorformat NV12\n");
    } else
      printf ("Invalid eglcolorformat %d\n", eglFrame.eglColorFormat);
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }
}

//const int max_width = 640;
//const int max_height = 480;

extern "C" void
init (CustomerFunction * pFuncs)
{
  pFuncs->fPreProcess = pre_process;
  pFuncs->fGPUProcess = gpu_process;
  pFuncs->fPostProcess = post_process;

  int max_width = atoi(getenv("CAM_WIDTH"));
  int max_height = atoi(getenv("CAM_HEIGHT"));

  /* Initialize maps from CPU */
  cv::Mat xmap(max_height, max_width, CV_32FC1);
  cv::Mat ymap(max_height, max_width, CV_32FC1);

  //fill matrices
  std::stringstream ss1(getenv("CAM_MAT"));
  cv::Mat cam(3, 3, cv::DataType<float>::type);
  for(int i=0;i<3;i++) for(int j=0;j<3;j++) ss1 >> cam.at<float>(i, j);

  std::stringstream ss2(getenv("CAM_DIS"));
  std::stringstream ss3(ss2.str());
  std::string oneWord;int count=0;while(ss2 >> oneWord) ++count;
  cv::Mat dist(count, 1, cv::DataType<float>::type);  
  for(int i=0;i<count;i++) ss3 >> dist.at<float>(i, 0);

  cv::fisheye::initUndistortRectifyMap(cam, dist, cv::Mat(), cam, cv::Size(max_width, max_height), CV_32FC1, xmap, ymap);

  /* upload to GpuMats */
  gpu_xmap.upload(xmap);
  gpu_ymap.upload(ymap);
}

extern "C" void
deinit (void)
{

}
