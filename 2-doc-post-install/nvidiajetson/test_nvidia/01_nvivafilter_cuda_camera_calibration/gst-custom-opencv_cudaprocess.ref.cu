#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#include "opencv2/core.hpp"
#include "opencv2/calib3d.hpp"
#include "opencv2/cudawarping.hpp" 

#include "cudaEGL.h"

#if defined(__cplusplus)
extern "C" void Handle_EGLImage (EGLImageKHR image);
extern "C" {
#endif

typedef enum {
  COLOR_FORMAT_Y8 = 0,
  COLOR_FORMAT_U8_V8,
  COLOR_FORMAT_RGBA,
  COLOR_FORMAT_NONE
} ColorFormat;

typedef struct {
  void (*fGPUProcess) (EGLImageKHR image, void ** userPtr);
  void (*fPreProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
  void (*fPostProcess)(void **sBaseAddr,
                      unsigned int *smemsize,
                      unsigned int *swidth,
                      unsigned int *sheight,
                      unsigned int *spitch,
                      ColorFormat *sformat,
                      unsigned int nsurfcount,
                      void ** userPtr);
} CustomerFunction;

void init (CustomerFunction * pFuncs);

#if defined(__cplusplus)
}
#endif


static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
   printf ("pre-process %dx%d size %d\n", *swidth, *sheight, *smemsize); 
}

static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
   printf ("post-process %dx%d size %d\n", *swidth, *sheight, *smemsize); 
}



static cv::cuda::GpuMat gpu_xmap, gpu_ymap;

static void cv_process_RGBA(void *pdata, int32_t width, int32_t height)
{
    cv::cuda::GpuMat d_Mat_RGBA(height, width, CV_8UC4, pdata);
    cv::cuda::GpuMat d_Mat_RGBA_Src;
    d_Mat_RGBA.copyTo(d_Mat_RGBA_Src); // cannot avoid one copy
    cv::cuda::remap(d_Mat_RGBA_Src, d_Mat_RGBA, gpu_xmap, gpu_ymap, cv::INTER_CUBIC, cv::BORDER_CONSTANT, cv::Scalar(0.f, 0.f, 0.f, 0.f));

    // Check
    if(d_Mat_RGBA.data != pdata)
	std::cerr << "Error reallocated buffer for d_Mat_RGBA" << std::endl;
}

static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);

  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }

  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_ABGR) {
 	cv_process_RGBA(eglFrame.frame.pPitch[0], eglFrame.width, eglFrame.height);
    } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
      printf ("Invalid eglcolorformat NV12\n");
    } else
      printf ("Invalid eglcolorformat %d\n", eglFrame.eglColorFormat);
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }
}

const int max_width = 640;
const int max_height = 480;

extern "C" void
init (CustomerFunction * pFuncs)
{
  pFuncs->fPreProcess = pre_process;
  pFuncs->fGPUProcess = gpu_process;
  pFuncs->fPostProcess = post_process;

  /* Initialize maps from CPU */
  cv::Mat xmap(max_height, max_width, CV_32FC1);
  cv::Mat ymap(max_height, max_width, CV_32FC1);

  //fill matrices
  cv::Mat cam(3, 3, cv::DataType<float>::type);
  cam.at<float>(0, 0) = 528.53618582196384f;
  cam.at<float>(0, 1) = 0.0f;
  cam.at<float>(0, 2) = 314.01736116032430f;

  cam.at<float>(1, 0) = 0.0f;
  cam.at<float>(1, 1) = 532.01912214324500f;
  cam.at<float>(1, 2) = 231.43930864205211f;

  cam.at<float>(2, 0) = 0.0f;
  cam.at<float>(2, 1) = 0.0f;
  cam.at<float>(2, 2) = 1.0f;

  cv::Mat dist(4, 1, cv::DataType<float>::type);  
  dist.at<float>(0, 0) = -0.11839989180635836f;
  dist.at<float>(1, 0) = 0.25425420873955445f;
  dist.at<float>(2, 0) = 0.0013269901775205413f;
  dist.at<float>(3, 0) = 0.0015787467748277866f;

  cv::fisheye::initUndistortRectifyMap(cam, dist, cv::Mat(), cam, cv::Size(max_width, max_height), CV_32FC1, xmap, ymap);

  /* upload to GpuMats */
  gpu_xmap.upload(xmap);
  gpu_ymap.upload(ymap);
}

extern "C" void
deinit (void)
{

}
